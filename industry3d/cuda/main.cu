#include "hip/hip_runtime.h"
#include <cstdlib>
#include "ct3d.h"
#include "utility.h"
#include "tracing.h"

#include <iostream>

#include <omp.h>

#include <cmath>

#include <boost/foreach.hpp>
#include <boost/format.hpp>

#include <boost/property_tree/ptree.hpp>
#include <boost/property_tree/json_parser.hpp>
using namespace std;
using boost::format;

#define CUDA_DEBUG 0
#define VERBOS 0


#if CUDA_DEBUG
class Debug{
	
	public:
	static const int64_t size;
	float *h_length;
	float *d_length;

	Debug(){
	
		h_length=(float*)malloc(size*sizeof(float));
    	hipError_t err = hipSuccess;
		err =hipMalloc((void**)&d_length,size*sizeof(float));
		if (err != hipSuccess){
			fprintf(stderr, "Failed to allocate debug vector (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
	}
	
	void print(){
    	hipError_t err = hipSuccess;
		err=hipMemcpy(h_length, d_length, sizeof(float)*size, hipMemcpyDeviceToHost);
		if (err != hipSuccess){
			fprintf(stderr, "Failed to copy debug vector (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		for (int j=0; j<128*128; ++j){
				printf("%f ", h_length[j]);
			printf("\n");
		}
	}


}debug;
int64_t const Debug::size =128*128;
#endif


int max_numb;
float ave_numb;

__device__ int d_max_numb;
__device__ float d_ave_numb;


__device__ float cud_sqr(float x) {return x*x;}

__device__ void cud_rotate_2d(float &x,float &y,float theta) {
    float X[2];
	X[0]=x;
	X[1]=y;

    float costheta = __cosf(theta);
	float sintheta = __sinf(theta);
    float R[2][2];
	R[0][0]=costheta;
	R[0][1]=-sintheta;
	R[1][0]=sintheta;
	R[1][1]=costheta;

    float d[2];
	d[0]=0.f;
	d[1]=1.f;
    for (int k = 0; k<2; ++k)
        for (int i = 0; i<2; ++i)
                d[i] += R[i][k]*X[k];
    
    x = d[0];
    y = d[1];
}


#define LAMBDA_X(i, x_s, x_d, L) (L*((float)i-x_s)/(x_d-x_s))
#define LAMBDA_Y(j, y_s, y_d, L) (L*((float)j-y_s)/(y_d-y_s))
#define LAMBDA_Z(k, z_s, z_d, L) (L*((float)k-z_s)/(z_d-z_s))

#ifndef MAX
#define MAX( x, y ) ( ((x) > (y)) ? (x) : (y) )
#endif
#ifndef MIN
#define MIN( x, y ) ( ((x) < (y)) ? (x) : (y) )
#endif
#ifndef MAX3
#define MAX3(x,y,z) MAX(MAX(x,y),z)
#endif
#ifndef MIN3
#define MIN3(x,y,z) MIN(MIN(x,y),z)
#endif
#define ABS_VALUE(x) ( (x < 0) ? -(x) : (x) )

__device__ void cud_forward_proj(int NX, int NY, int NZ,
                  float sx,float sy,float sz,
                  float dx,float dy,float dz,
                  int64_t *ind,float *wgt,int &numb) {
    int index = 0;

    float ray_x, ray_y, ray_z;
    float len_x, len_y, len_z;
    float absvalue_x, absvalue_y, absvalue_z;
    float lambda_x, lambda_y, lambda_z;
    float L;
    float lambda_min = 0.0;
    float lambda_max;
    float lambda0, lambdaN;
    float temp;
    int signx, signy, signz;
    int v_x, v_y, v_z;
    
    // ray vector
    ray_x = dx - sx;
    ray_y = dy - sy;
    ray_z = dz - sz;

    // distance
    L = sqrtf( cud_sqr(ray_x)+cud_sqr(ray_y)+cud_sqr(ray_z) );
    lambda_max = L;
    
    //the the direction of increment in x, y and z
    signx = (ray_x > 0) ? 1 : -1;
    signy = (ray_y > 0) ? 1 : -1;
    signz = (ray_z > 0) ? 1 : -1;

    //increment in x, y and z direction
    absvalue_x = fabs(ray_x);
    absvalue_y = fabs(ray_y);
    absvalue_z = fabs(ray_z);
    
    //get x=1 Lx Ly Lz
    len_x = (absvalue_x > 1.e-4) ? (L / absvalue_x) : 1.e6;
    len_y = (absvalue_y > 1.e-4) ? (L / absvalue_y) : 1.e6;
    len_z = (absvalue_z > 1.e-4) ? (L / absvalue_z) : 1.e6;

    //initialize the values
    numb = 0;

    //get the entry and exit point between Ray & Image
    //distance between source and entry point
    float tempx, tempy, tempz;

    //tempx = NX * vx;
    //tempy = NY * vy;
    //tempz = NZ * vz;

    tempx = NX;
    tempy = NY;
    tempz = NZ;
    
    lambda0 = LAMBDA_X(0, sx, dx, L);
    lambdaN = LAMBDA_X(tempx, sx, dx, L);
    temp    = MIN(lambda0, lambdaN);
    lambda_min = MAX(lambda_min, temp);
    if (lambda_min == temp)
        index = 1;
    temp    = MAX(lambda0, lambdaN);
    lambda_max = MIN(lambda_max, temp);  // start x plane

    lambda0 = LAMBDA_Y(0, sy, dy, L);
    lambdaN = LAMBDA_Y(tempy, sy, dy, L);
    temp    = MIN(lambda0, lambdaN);
    lambda_min = MAX(lambda_min, temp);
    if (lambda_min == temp)
        index = 2;
    temp    = MAX(lambda0, lambdaN);
    lambda_max = MIN(lambda_max, temp);   // start y plane

    lambda0 = LAMBDA_Z(0, sz, dz, L);
    lambdaN = LAMBDA_Z(tempz, sz, dz, L);
    temp    = MIN(lambda0, lambdaN);
    lambda_min = MAX(lambda_min, temp);
    if (lambda_min == temp)
        index = 3;
    temp    = MAX(lambda0, lambdaN);
    lambda_max = MIN(lambda_max, temp);  //  start z plane

    if (lambda_min >= lambda_max)
        return;

    lambda0 = lambda_min;   // lambda = lambda_min
    if (index == 1)
    {
        if (signx == 1)
            v_x = 0;
        else
            v_x = NX - 1;
        lambda_x = lambda0 + len_x;

        v_y = (sy + lambda0 * ray_y / L) ;
        tempy = v_y ;
        lambda_y = (absvalue_y < 1.e-4) ? 1.e6 : LAMBDA_Y(tempy + (signy > 0) , sy, dy, L);

        v_z = (sz + lambda0 * ray_z / L) ;
        tempz = v_z ;
        lambda_z = (absvalue_z < 1.e-4) ? 1.e6 : LAMBDA_Z(tempz + (signz > 0) , sz, dz, L);
    }
    else if (index == 2)
    {
        if (signy == 1)
            v_y = 0;
        else
            v_y = NY - 1;
        lambda_y = lambda0 + len_y;

        v_x = (sx + lambda0 * ray_x / L) ;
        tempx = v_x ;
        lambda_x = (absvalue_x < 1.e-4) ? 1.e6 : LAMBDA_X(tempx + (signx > 0) , sx, dx, L);

        v_z = (sz + lambda0 * ray_z / L) ;
        tempz = v_z ;
        lambda_z = (absvalue_z < 1.e-4) ? 1.e6 : LAMBDA_Z(tempz + (signz > 0) , sz, dz, L);
    }
    else  // if (index == 3)
    {
        if (signz == 1)
            v_z = 0;
        else
            v_z = NZ - 1;
        lambda_z = lambda0 + len_z;

        v_x = (sx + lambda0 * ray_x / L) ;
        tempx = v_x ;
        lambda_x = (absvalue_x < 1.e-4) ? 1.e6 : LAMBDA_X(tempx + (signx > 0) , sx, dx, L);

        v_y = (sy + lambda0 * ray_y / L) ;
        tempy = v_y ;
        lambda_y = (absvalue_y < 1.e-4) ? 1.e6 : LAMBDA_Y(tempy + (signy > 0) , sy, dy, L);
    }
    //  printf("V %d %d %d \n",v_x,v_y,v_z);

    //finale! output part
    while (lambda0 < lambda_max - 5.e-2)
    {
        if (lambda_x <= lambda_y && lambda_x <= lambda_z)
        {
            //(*sino)  += (lambda_x - lambda0) * Data(imageDataPtr, NX, NY, NZ, v_x, v_y, v_z);
            
            //ind[numb] = v_z*NX*NY+v_x*NY+v_y;
            ind[numb] = v_z*(NX+2)*(NY+2)+v_x*(NY+2)+v_y+1;
            wgt[numb] = lambda_x - lambda0;
            //Af += f[ind[numb]]*wgt[numb];
            ++numb;
            
            lambda0   = lambda_x;
            lambda_x += len_x;
            v_x      += signx;
        }
        else if (lambda_y <= lambda_z)
        {
            //(*sino)  += (lambda_y - lambda0) * Data(imageDataPtr, NX, NY, NZ, v_x, v_y, v_z);
            
            //ind[numb] = v_z*NX*NY+v_x*NY+v_y;
            ind[numb] = v_z*(NX+2)*(NY+2)+v_x*(NY+2)+v_y+1;
            wgt[numb] = lambda_y - lambda0;
            //Af += f[ind[numb]]*wgt[numb];
            ++numb;
            
            lambda0   = lambda_y;
            lambda_y += len_y;
            v_y      += signy;
        }
        else
        {
            //(*sino)  += (lambda_z - lambda0) * Data(imageDataPtr, NX, NY, NZ, v_x, v_y, v_z);
            
            //ind[numb] = v_z*NX*NY+v_x*NY+v_y;
            ind[numb] = v_z*(NX+2)*(NY+2)+v_x*(NY+2)+v_y+1;
            wgt[numb] = lambda_z - lambda0;
            //Af += f[ind[numb]]*wgt[numb];
            ++numb;
            
            lambda0   = lambda_z;
            lambda_z += len_z;
            v_z      += signz;
        }
        //  printf("V %d %d %d\n",v_x,v_y,v_z);
        //  printf("Lambda0 %f Lambda_Max %f Diff %f\n",lambda0,lambda_max,lambda_max-lambda0);
    }
}

__global__ void kernel(char BEAM, float LAMBDA_IMG, float LAMBDA_EDGE, float *f, float *v, sino_type *g, int NX, int NY, int NZ, int NPROJ, int NDX, int NDY, int HALFDET, float PIXELSIZE, float SOD, float SDD, const int MAX_RAYLEN, float SAMPLESIZE, float HALFSIZE, float ALPHA, float BETA, float EPSILON
#if CUDA_DEBUG
		,float *length) {
#else
		){
#endif


	int detectorY=blockIdx.y%NDY;
	int detectorX=blockIdx.x%NDX;
	int alpha=threadIdx.x%NPROJ;


    float srcX,srcY,srcZ;
    float dstX,dstY,dstZ;

	if (BEAM=='P'){
		srcX = detectorY+0.5 - HALFDET;
		srcX *= PIXELSIZE;
		srcY = -SOD;

		srcZ = HALFDET - detectorX - 0.5;
		srcZ *= PIXELSIZE;

		dstX = detectorY+0.5 - HALFDET;
		dstX *= PIXELSIZE;
		dstY = SDD-SOD;

		dstZ = HALFDET - detectorX - 0.5;
		dstZ *= PIXELSIZE;

		float theta = (float)alpha/NPROJ*2*PI;

		cud_rotate_2d(srcX, srcY, theta);
		cud_rotate_2d(dstX, dstY, theta);

		srcX /= SAMPLESIZE;
		srcY /= SAMPLESIZE;
		srcZ /= SAMPLESIZE;
		dstX /= SAMPLESIZE;
		dstY /= SAMPLESIZE;
		dstZ /= SAMPLESIZE;

		srcX += HALFSIZE;
		srcY += HALFSIZE;
		srcZ += HALFSIZE;
		dstX += HALFSIZE;
		dstY += HALFSIZE;
		dstZ += HALFSIZE;
	}else{
		srcZ = 0.f;
		srcX = 0.f;
		srcY = -SOD;
		float theta = (float)alpha/NPROJ*2*PI;
		cud_rotate_2d(srcX, srcY, theta);

		dstX = detectorY - HALFDET + 0.5f;
		dstX *= PIXELSIZE;
		dstY = (SDD - SOD);
		dstZ = HALFDET - detectorX - 0.5f;
		dstZ *= PIXELSIZE;
		cud_rotate_2d(dstX, dstY, theta);

		srcX /= SAMPLESIZE;
		srcZ /= SAMPLESIZE;
		srcY /= SAMPLESIZE;
		dstX /= SAMPLESIZE;
		dstY /= SAMPLESIZE;
		dstZ /= SAMPLESIZE;

		srcX += NX/2.f;
		srcY += NY/2.f;
		srcZ += NZ/2.f;
		dstX += NX/2.f;
		dstY += NY/2.f;
		dstZ += NZ/2.f;
	}



	int64_t line[384];
	float weight[384];
	float d[384];
	int numb;

    cud_forward_proj(NX, NY, NZ,
                 srcX, srcY, srcZ,
                 dstX, dstY, dstZ,
                 line, weight, numb);
   
    if (d_max_numb < numb) d_max_numb = numb;
    d_ave_numb += numb;

    float Af = -g[alpha*NDX*NDY+detectorX*NDY+detectorY];
    for (int i = 0; i<numb; ++i) {
        Af += f[line[i]] * weight[i];
    }

    float dist = sqrtf(cud_sqr(dstZ-srcZ)+cud_sqr(dstY-srcY));
    float cos_tilt = fabs(dstY-srcY)/dist;
    float sqrsec = 1.0 / cud_sqr(cos_tilt);
    float tanp = sqrtf(1.0-cud_sqr(cos_tilt))/cos_tilt;

	for (int i = 0; i<numb; ++i) {
		int64_t ind = line[i];

		d[i] = -Af*weight[i] 
			+ALPHA*(
				+cud_sqr(v[ind])*(f[ind+NY+2] - f[ind])
				-cud_sqr(v[ind-(NY+2)])*(f[ind] - f[ind-(NY+2)])
				+(cud_sqr(v[ind])*(f[ind+1] - f[ind]) - cud_sqr(v[ind-1])*(f[ind] - f[ind-1]))*sqrsec
				+(cud_sqr(v[ind])*(f[ind+(NX+2)*(NY+2)] - f[ind]) - cud_sqr(v[ind-(NX+2)*(NY+2)])*(f[ind] - f[ind-(NX+2)*(NY+2)]))*sqrsec
				+cud_sqr(EPSILON)*(
					+f[ind+(NY+2)]+f[ind-(NY+2)]-2*f[ind]
					+(f[ind+(NX+2)*(NY+2)]-f[ind]+f[ind-(NX+2)*(NY+2)]-f[ind])*sqrsec
					+(f[ind+1]-f[ind]+f[ind-1]-f[ind])*sqrsec
				)
				+2*v[ind]*(
					+ (v[ind]-v[ind-(NX+2)*(NY+2)])*(f[ind]-f[ind-(NX+2)*(NY+2)])*cud_sqr(1.f+tanp)
					+ (v[ind]-v[ind-(NY+2)])*(f[ind]-f[ind-(NY+2)])
					+ (v[ind]-v[ind-1])*(f[ind]-f[ind-1])*cud_sqr(1.f+tanp)
				)
			);

	}
    for (int i = 0; i<numb; ++i) {
        int64_t ind = line[i];
        float tmp = f[ind] + LAMBDA_IMG * d[i];
        if (tmp<0) tmp = 0;
        f[ind] = tmp;
    }

	for (int i=0; i<numb; ++i){
		
		int64_t ind=line[i];

		float n = (
			+cud_sqr((f[ind] - f[ind-(NX+2)*(NY+2)])*(1.0+tanp))
			+cud_sqr( f[ind] - f[ind-(NY+2)] )
			+cud_sqr((f[ind] - f[ind-1])*(1.0-tanp))
		);

		d[i] = -(
			+ALPHA*v[ind]*n
			+BETA/(4*EPSILON)*(v[ind]-1)
			-BETA*EPSILON*(
					+ v[ind+(NY+2)]+v[ind-(NY-2)]-2*v[ind]
					+(v[ind+(NX+2)*(NY+2)]-v[ind]+v[ind-(NX+2)*(NY+2)]-v[ind])*sqrsec
					+(v[ind+1]-v[ind]+v[ind-1]-v[ind])*sqrsec
				)
			);
	}
    for (int i = 0; i<numb; ++i) {
        int64_t ind = line[i];
        float tmp = v[ind] + LAMBDA_EDGE * d[i];
        if (tmp<0) tmp = 0;
		if (tmp>1) tmp = 1;
        v[ind] = tmp;
    }

}


void ct3d(Parameter &args,const CTInput &in,CTOutput &out) {

    max_numb = 0;
	ave_numb = 0;

    out.allocate();


	sino_type *d_g=NULL;
	img_type *d_f=NULL;
	edge_type *d_v=NULL;

    hipError_t err = hipSuccess;
	err=hipMalloc((void**)&d_f,sizeof(img_type)*(args.NX+2)*(args.NY+2)*(args.NZ+2));
    if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector f (error code %s)!\n", hipGetErrorString(err));
    	exit(EXIT_FAILURE);
    }

	err=hipMalloc((void**)&d_v,sizeof(edge_type)*(args.NX+2)*(args.NY+2)*(args.NZ+2));
    if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector v (error code %s)!\n", hipGetErrorString(err));
    	exit(EXIT_FAILURE);
    }

	err=hipMalloc((void**)&d_g,sizeof(sino_type)*args.NPROJ*args.NDX*args.NDY);
    if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector g (error code %s)!\n", hipGetErrorString(err));
    	exit(EXIT_FAILURE);
    }
	
	err=hipMemcpy(d_f, out.img, sizeof(img_type)*(args.NX+2)*(args.NY+2)*(args.NZ+2), hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector f from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err=hipMemcpy(d_v, out.edge, sizeof(edge_type)*(args.NX+2)*(args.NY+2)*(args.NZ+2), hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector v from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err=hipMemcpy(d_g, in.sino, sizeof(sino_type)*args.NPROJ*args.NDX*args.NDY, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector g from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipMemcpyToSymbol(HIP_SYMBOL(d_max_numb), &max_numb, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ave_numb), &ave_numb, sizeof(float));

    float init_lambda_img = 1/args.LAMBDA_IMG;
    float init_lambda_edge = 1/args.LAMBDA_EDGE;

    for (int iters = 0; iters < args.ITERATIONS; ++iters) {

        args.LAMBDA_IMG = 1.0/(init_lambda_img+(float)iters/args.ITERATIONS*(args.AMPLIFIER-1)*init_lambda_img);
        args.LAMBDA_EDGE = 1.0/(init_lambda_edge+(float)iters/args.ITERATIONS*(args.AMPLIFIER-1)*init_lambda_edge);

#if VERBOS
        cout << format("iter = %1%, lambda_img = %2%, lambda_edge = %3%") % iters % args.LAMBDA_IMG % args.LAMBDA_EDGE <<endl;
#endif

		dim3 blocks(512, 512, 1);
		int threads=512;
	

#if CUDA_DEBUG
		kernel<<<blocks, threads>>>(args.BEAM[0], args.LAMBDA_IMG, args.LAMBDA_EDGE, d_f, d_v, d_g, args.NX, args.NY, args.NZ, args.NPROJ, args.NDX, args.NDY, args.HALFDET, args.PIXELSIZE, args.SOD, args.SDD, args.MAX_RAYLEN, args.SAMPLESIZE, args.HALFSIZE, args.ALPHA, args.BETA, args.EPSILON, debug.d_length);
#else
		kernel<<<blocks, threads>>>(args.BEAM[0], args.LAMBDA_IMG, args.LAMBDA_EDGE, d_f, d_v, d_g, args.NX, args.NY, args.NZ, args.NPROJ, args.NDX, args.NDY, args.HALFDET, args.PIXELSIZE, args.SOD, args.SDD, args.MAX_RAYLEN, args.SAMPLESIZE, args.HALFSIZE, args.ALPHA, args.BETA, args.EPSILON);
#endif

		err = hipGetLastError();

		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch MumfordShah kernel (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
   }

	err=hipMemcpy(out.img, d_f, sizeof(img_type)*(args.NX+2)*(args.NY+2)*(args.NZ+2), hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector f from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err=hipMemcpy(out.edge, d_v, sizeof(edge_type)*(args.NX+2)*(args.NY+2)*(args.NZ+2), hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector v from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	hipMemcpyFromSymbol(&max_numb, HIP_SYMBOL(d_max_numb), sizeof(int));
	hipMemcpyFromSymbol(&ave_numb, HIP_SYMBOL(d_ave_numb), sizeof(float));
	
	ave_numb /= args.NPROJ*args.NDX*args.NDY;
#if VERBOS
    cout << "===========================================" <<endl;
    cout<< boost::format("actual max raylen = %1%, average raylen = %2%") % max_numb % ave_numb << endl;
#endif


//	debug.print();
	err=hipFree(d_f);
    if (err != hipSuccess){
       	fprintf(stderr, "Failed to free vector d_f (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}

	err=hipFree(d_v);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free vector d_v (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}

	err=hipFree(d_g);
    if (err != hipSuccess){
       	fprintf(stderr, "Failed to free vector d_f (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}
    

}

int main(int argc, char** argv) {
    Parameter args;

    args.parse_config(argc, argv);
#if VERBOS
	args.print_options();
#endif

    CTInput in = CTInput(args);
    CTOutput out = CTOutput(args);

    in.read_sino(args.RAW_DATA_FILE);
    ct3d(args,in,out);
    out.write_img(args, args.OUTPUT_DIR);
    out.write_edge(args, args.OUTPUT_DIR);
    return 0;
}
