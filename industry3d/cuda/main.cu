#include "hip/hip_runtime.h"
#include "ct3d.h"
#include "utility.h"
#include "tracing.h"

#include <iostream>

#include <omp.h>

#include <cmath>

#include <boost/foreach.hpp>
#include <boost/format.hpp>

#include <boost/property_tree/ptree.hpp>
#include <boost/property_tree/json_parser.hpp>
using namespace std;
using boost::format;


class Debug{
	
	public:
	static const long size;
	
	int *h_length;
	int *d_length;

	Debug(){
	
		h_length=(int*)malloc(size*sizeof(int));
    	hipError_t err = hipSuccess;
		err =hipMalloc((void**)&d_length,size*sizeof(int));
		if (err != hipSuccess){
			fprintf(stderr, "Failed to allocate debug vector (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

	}
	
	void transferToHost(){
    	hipError_t err = hipSuccess;
		err=hipMemcpy(h_length, d_length, sizeof(int)*size, hipMemcpyDeviceToHost);
		if (err != hipSuccess){
			fprintf(stderr, "Failed to copy debug vector (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

	}

	void print(){
			for (int j=0; j<128*128; ++j)
				for (int k=0; k<384; ++k)
					printf("%d ", h_length[j*384+k]);
	}


};

long const Debug::size = 128*128*384;


int max_numb;
float ave_numb;

__device__ int d_max_numb;
__device__ float d_ave_numb;

__device__ float cud_sqr(float x) {return x*x;}


__device__ void cud_rotate_2d(float &x,float &y,float theta) {
    float X[2];
	X[0]=x;
	X[1]=y;

    float costheta = __cosf(theta);
	float sintheta = __sinf(theta);
    float R[2][2];
	R[0][0]=costheta;
	R[0][1]=-sintheta;
	R[1][0]=sintheta;
	R[1][1]=costheta;

    float d[2];
	d[0]=0.f;
	d[1]=1.f;
    for (int k = 0; k<2; ++k)
        for (int i = 0; i<2; ++i)
                d[i] += R[i][k]*X[k];
    
    x = d[0];
    y = d[1];
}


#define LAMBDA_X(i, x_s, x_d, L) (L*((float)i-x_s)/(x_d-x_s))
#define LAMBDA_Y(j, y_s, y_d, L) (L*((float)j-y_s)/(y_d-y_s))
#define LAMBDA_Z(k, z_s, z_d, L) (L*((float)k-z_s)/(z_d-z_s))

#ifndef MAX
#define MAX( x, y ) ( ((x) > (y)) ? (x) : (y) )
#endif
#ifndef MIN
#define MIN( x, y ) ( ((x) < (y)) ? (x) : (y) )
#endif
#ifndef MAX3
#define MAX3(x,y,z) MAX(MAX(x,y),z)
#endif
#ifndef MIN3
#define MIN3(x,y,z) MIN(MIN(x,y),z)
#endif
#define ABS_VALUE(x) ( (x < 0) ? -(x) : (x) )

__device__ int64_t cud_get_img_addr(const Parameter &args,int x,int y,int z) {
    return (int64_t)z*args.NX*args.NY+x*args.NY+y;
}

__device__ void cud_forward_proj(int NX, int NY, int NZ,
                  float sx,float sy,float sz,
                  float dx,float dy,float dz,
                  int64_t *ind,float *wgt,int &numb) {

    int index = 0;

    float ray_x, ray_y, ray_z;
    float len_x, len_y, len_z;
    float absvalue_x, absvalue_y, absvalue_z;
    float lambda_x, lambda_y, lambda_z;
    float L;
    float lambda_min = 0.0;
    float lambda_max;
    float lambda0, lambdaN;
    float temp;
    int signx, signy, signz;
    int v_x, v_y, v_z;
    
    // ray vector
    ray_x = dx - sx;
    ray_y = dy - sy;
    ray_z = dz - sz;

    // distance
    L = sqrt( cud_sqr(ray_x)+cud_sqr(ray_y)+cud_sqr(ray_z) );
    lambda_max = L;
    
    //the the direction of increment in x, y and z
    signx = (ray_x > 0) ? 1 : -1;
    signy = (ray_y > 0) ? 1 : -1;
    signz = (ray_z > 0) ? 1 : -1;

    //increment in x, y and z direction
    absvalue_x = fabs(ray_x);
    absvalue_y = fabs(ray_y);
    absvalue_z = fabs(ray_z);
    
    //get x=1 Lx Ly Lz
    len_x = (absvalue_x > 1.e-4) ? (L / absvalue_x) : 1.e6;
    len_y = (absvalue_y > 1.e-4) ? (L / absvalue_y) : 1.e6;
    len_z = (absvalue_z > 1.e-4) ? (L / absvalue_z) : 1.e6;

    //initialize the values
    numb = 0;

    //get the entry and exit point between Ray & Image
    //distance between source and entry point
    float tempx, tempy, tempz;

    //tempx = NX * vx;
    //tempy = NY * vy;
    //tempz = NZ * vz;

    tempx = NX;
    tempy = NY;
    tempz = NZ;
    
    lambda0 = LAMBDA_X(0, sx, dx, L);
    lambdaN = LAMBDA_X(tempx, sx, dx, L);
    temp    = MIN(lambda0, lambdaN);
    lambda_min = MAX(lambda_min, temp);
    if (lambda_min == temp)
        index = 1;
    temp    = MAX(lambda0, lambdaN);
    lambda_max = MIN(lambda_max, temp);  // start x plane

    lambda0 = LAMBDA_Y(0, sy, dy, L);
    lambdaN = LAMBDA_Y(tempy, sy, dy, L);
    temp    = MIN(lambda0, lambdaN);
    lambda_min = MAX(lambda_min, temp);
    if (lambda_min == temp)
        index = 2;
    temp    = MAX(lambda0, lambdaN);
    lambda_max = MIN(lambda_max, temp);   // start y plane

    lambda0 = LAMBDA_Z(0, sz, dz, L);
    lambdaN = LAMBDA_Z(tempz, sz, dz, L);
    temp    = MIN(lambda0, lambdaN);
    lambda_min = MAX(lambda_min, temp);
    if (lambda_min == temp)
        index = 3;
    temp    = MAX(lambda0, lambdaN);
    lambda_max = MIN(lambda_max, temp);  //  start z plane

    if (lambda_min >= lambda_max)
        return;
    lambda0 = lambda_min;   // lambda = lambda_min
    if (index == 1)
    {
        if (signx == 1)
            v_x = 0;
        else
            v_x = NX - 1;
        lambda_x = lambda0 + len_x;

        v_y = (sy + lambda0 * ray_y / L) ;
        tempy = v_y ;
        lambda_y = (absvalue_y < 1.e-4) ? 1.e6 : LAMBDA_Y(tempy + (signy > 0) , sy, dy, L);

        v_z = (sz + lambda0 * ray_z / L) ;
        tempz = v_z ;
        lambda_z = (absvalue_z < 1.e-4) ? 1.e6 : LAMBDA_Z(tempz + (signz > 0) , sz, dz, L);
    }
    else if (index == 2)
    {
        if (signy == 1)
            v_y = 0;
        else
            v_y = NY - 1;
        lambda_y = lambda0 + len_y;

        v_x = (sx + lambda0 * ray_x / L) ;
        tempx = v_x ;
        lambda_x = (absvalue_x < 1.e-4) ? 1.e6 : LAMBDA_X(tempx + (signx > 0) , sx, dx, L);

        v_z = (sz + lambda0 * ray_z / L) ;
        tempz = v_z ;
        lambda_z = (absvalue_z < 1.e-4) ? 1.e6 : LAMBDA_Z(tempz + (signz > 0) , sz, dz, L);
    }
    else  // if (index == 3)
    {
        if (signz == 1)
            v_z = 0;
        else
            v_z = NZ - 1;
        lambda_z = lambda0 + len_z;

        v_x = (sx + lambda0 * ray_x / L) ;
        tempx = v_x ;
        lambda_x = (absvalue_x < 1.e-4) ? 1.e6 : LAMBDA_X(tempx + (signx > 0) , sx, dx, L);

        v_y = (sy + lambda0 * ray_y / L) ;
        tempy = v_y ;
        lambda_y = (absvalue_y < 1.e-4) ? 1.e6 : LAMBDA_Y(tempy + (signy > 0) , sy, dy, L);
    }
    //  printf("V %d %d %d \n",v_x,v_y,v_z);

    //finale! output part
    while (lambda0 < lambda_max - 5.e-2)
    {
        if (lambda_x <= lambda_y && lambda_x <= lambda_z)
        {
            //(*sino)  += (lambda_x - lambda0) * Data(imageDataPtr, NX, NY, NZ, v_x, v_y, v_z);
            
    		ind[numb] =  (int64_t)v_z*NX*NY+v_x*NY+v_y;
            wgt[numb] = lambda_x - lambda0;
            //Af += f[ind[numb]]*wgt[numb];
            ++numb;
            
            lambda0   = lambda_x;
            lambda_x += len_x;
            v_x      += signx;
        }
        else if (lambda_y <= lambda_z)
        {
            //(*sino)  += (lambda_y - lambda0) * Data(imageDataPtr, NX, NY, NZ, v_x, v_y, v_z);
            
    		ind[numb] =  (int64_t)v_z*NX*NY+v_x*NY+v_y;
            wgt[numb] = lambda_y - lambda0;
            //Af += f[ind[numb]]*wgt[numb];
            ++numb;
            
            lambda0   = lambda_y;
            lambda_y += len_y;
            v_y      += signy;
        }
        else
        {
            //(*sino)  += (lambda_z - lambda0) * Data(imageDataPtr, NX, NY, NZ, v_x, v_y, v_z);
            
    		ind[numb] =  (int64_t)v_z*NX*NY+v_x*NY+v_y;
            wgt[numb] = lambda_z - lambda0;
            //Af += f[ind[numb]]*wgt[numb];
            ++numb;
            
            lambda0   = lambda_z;
            lambda_z += len_z;
            v_z      += signz;
        }
        //  printf("V %d %d %d\n",v_x,v_y,v_z);
        //  printf("Lambda0 %f Lambda_Max %f Diff %f\n",lambda0,lambda_max,lambda_max-lambda0);
    }
}

__global__ void kernel(float LAMBDA_IMG, float LAMBDA_EDGE, float *f, float *v, float *g, int NX, int NY, int NZ, int NPROJ, int NDX, int NDY, int HALFDET, float PIXELSIZE, float SOD, float SDD, const int MAX_RAYLEN, float SAMPLESIZE, float HALFSIZE, float ALPHA, float BETA, float EPSILON,
		int *length) {


	int detectorX=blockIdx.x%NDX;
	//int alpha=blockIdx.y%NPROJ;
	int alpha=64;
	int detectorY=threadIdx.x%NDY;

    float srcX,srcY,srcZ;
    float dstX,dstY,dstZ;

	//cud_parallel

    srcX = detectorY+0.5 - HALFDET;
    srcX *= PIXELSIZE;
    srcY = -SOD;

    srcZ = HALFDET - detectorX - 0.5;
    srcZ *= PIXELSIZE;

    dstX = detectorY+0.5 - HALFDET;
    dstX *= PIXELSIZE;
    dstY = SDD-SOD;

    dstZ = HALFDET - detectorX - 0.5;
    dstZ *= PIXELSIZE;

    float theta = (float)alpha/NPROJ*2*PI;

    cud_rotate_2d(srcX, srcY, theta);
    cud_rotate_2d(dstX, dstY, theta);

    srcX /= SAMPLESIZE;
    srcY /= SAMPLESIZE;
    srcZ/= SAMPLESIZE;
    dstX /= SAMPLESIZE;
    dstY /= SAMPLESIZE;
    dstZ /= SAMPLESIZE;

    srcX += HALFSIZE;
    srcY += HALFSIZE;
    srcZ += HALFSIZE;
    dstX += HALFSIZE;
    dstY += HALFSIZE;
    dstZ += HALFSIZE;


	/*
	//cud_cone
    srcZ = 0.0;
    srcX = 0.0;
    srcY = -SOD;
    float theta = (float)alpha/NPROJ*2*PI;
    cud_rotate_2d(srcX, srcY, theta);

    dstX = detectorY - HALFDET + 0.5;
    dstX *= PIXELSIZE;
    dstY = (SDD - SOD);
    dstZ = HALFDET - detectorX - 0.5;
    dstZ *= PIXELSIZE;
    cud_rotate_2d(dstX, dstY, theta);

    srcX /= SAMPLESIZE;
    srcZ /= SAMPLESIZE;
    srcY /= SAMPLESIZE;
    dstX /= SAMPLESIZE;
    dstY /= SAMPLESIZE;
    dstZ /= SAMPLESIZE;

    srcX += NX/2.0;
    srcY += NY/2.0;
    srcZ += NZ/2.0;
    dstX += NX/2.0;
    dstY += NY/2.0;
    dstZ += NZ/2.0;
	*/

    float Af = -g[alpha*NDX*NDY+detectorX*NDY+detectorY];

	int64_t line[384];
	float weight[384];
	float d[384];
    int numb;

	for (int i=0; i<MAX_RAYLEN; ++i){
		d[i]=0.f;
		line[i]=0;
		weight[i]=0.f;
		numb = 0;
	}

    cud_forward_proj(NX, NY, NZ,
                 srcX, srcY, srcZ,
                 dstX, dstY, dstZ,
                 line, weight, numb);
   
    if (d_max_numb < numb) d_max_numb = numb;
    d_ave_numb += numb;


	length[detectorX*NDY+detectorY]=numb;
	return;

    for (int i = 0; i<numb; ++i) {
        Af += f[line[i]] * weight[i];
    }
    for (int i = 0; i<numb; ++i) {
        int64_t ind = line[i];
        int64_t plain = ind%(NX*NY);
        int x = plain/NY, y = plain%NY;

        float tmp = 0.;
        float lap = 0.;
        
        if (x+1<NX) tmp += cud_sqr(v[ind])*(v[ind+NY]-v[ind]);
        else             tmp += cud_sqr(v[ind])*(       0        -v[ind]);
        
        if (y+1<NY) tmp += cud_sqr(v[ind])*(v[ind+1]-v[ind]);
        else             tmp += cud_sqr(v[ind])*(     0    -v[ind]);
        
        if (x-1>=0)      tmp -= cud_sqr(v[ind-NY])*(v[ind]-v[ind-NY]);
        else             tmp -=                        (v[ind]-0        );
        
        if (y-1>=0)      tmp -= cud_sqr(v[ind-1])*(v[ind]-v[ind-1]);
        else             tmp -=                  (v[ind]-0       );
        
        if (x+1<NX) lap += v[ind+NY];
        if (y+1<NY) lap += v[ind+1];
        if (x-1>=0)      lap += v[ind-NY];
        if (y-1>=0)      lap += v[ind-1];
        lap -= 4*v[ind];

        d[i] = -Af*weight[i]+ALPHA*(tmp+cud_sqr(EPSILON)*lap);
    }
    for (int i = 0; i<numb; i++) {
        int64_t ind = line[i];
        float tmp = f[ind] + LAMBDA_IMG * d[i];
        if (tmp<0) tmp = 0;
        f[ind] = tmp;
    }

	for (int i=0; i<MAX_RAYLEN; ++i)
		d[i]=0.f;
    for (int i = 0; i<numb; ++i) {
        int64_t ind = line[i];
        int64_t plain = ind%(NX*NY);
        int x = plain/NY, y = plain%NY;

        float a = 0.;
        float b = 0.;
        float c = 0.;

        if (x-1>=0)      a += cud_sqr(v[ind]-v[ind-NY]);
        else             a += cud_sqr(v[ind]-0        );
        
        if (y-1>=0)      a += cud_sqr(v[ind]-v[ind-1]);
        else             a += cud_sqr(v[ind]-0       );
        
        a *= v[ind];
        
        b = v[ind]-1;

        if (x+1<NX) c += v[ind+NY];
        if (y+1<NY) c += v[ind+1];
        if (x-1>=0)      c += v[ind-NY];
        if (y-1>=0)      c += v[ind-1];
        c -= 4*v[ind];
        
        d[i] = -ALPHA*a-BETA/(4*EPSILON)*b+BETA*EPSILON*c;
    }
    for (int i = 0; i<numb; i++) {
        int64_t ind = line[i];
        float tmp = v[ind] + LAMBDA_EDGE * d[i];
        if (tmp<0) tmp = 0;
        if (tmp>1) tmp = 1;
        v[ind] = tmp;
    }
}


void ct3d(Parameter &args,const CTInput &in,CTOutput &out) {

    max_numb = 0;
	ave_numb = 0;

    out.allocate();

    int64_t global_start = timer_s();


	float *d_g=NULL;
	float *d_f=NULL;
	float *d_v=NULL;

    hipError_t err = hipSuccess;
	err=hipMalloc((void**)&d_f,sizeof(float)*args.NX*args.NY*args.NZ);
    if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector f (error code %s)!\n", hipGetErrorString(err));
    	exit(EXIT_FAILURE);
    }

	err=hipMalloc((void**)&d_v,sizeof(float)*args.NX*args.NY*args.NZ);
    if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector v (error code %s)!\n", hipGetErrorString(err));
    	exit(EXIT_FAILURE);
    }

	err=hipMalloc((void**)&d_g,sizeof(float)*args.NPROJ*args.NDX*args.NDY);
    if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector g (error code %s)!\n", hipGetErrorString(err));
    	exit(EXIT_FAILURE);
    }
	
	err=hipMemcpy(d_f, out.img, sizeof(float)*args.NX*args.NY*args.NZ, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector f from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err=hipMemcpy(d_v, out.edge, sizeof(float)*args.NX*args.NY*args.NZ, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector v from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err=hipMemcpy(d_g, in.sino, sizeof(float)*args.NPROJ*args.NDX*args.NDY, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector g from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipMemcpyToSymbol(HIP_SYMBOL(d_max_numb), &max_numb, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_ave_numb), &ave_numb, sizeof(float));

    float init_lambda_img = 1/args.LAMBDA_IMG;
    float init_lambda_edge = 1/args.LAMBDA_EDGE;
	Debug debug;

    for (int iters = 0; iters < args.ITERATIONS; ++iters) {

        args.LAMBDA_IMG = 1.0/(init_lambda_img+(float)iters/args.ITERATIONS*(args.AMPLIFIER-1)*init_lambda_img);
        args.LAMBDA_EDGE = 1.0/(init_lambda_edge+(float)iters/args.ITERATIONS*(args.AMPLIFIER-1)*init_lambda_edge);
        cout << format("iter = %1%, lambda_img = %2%, lambda_edge = %3%") % iters % args.LAMBDA_IMG % args.LAMBDA_EDGE <<endl;

		dim3 blocks(128, 1, 1);
		int threads=128;


		kernel<<<blocks, threads>>>(args.LAMBDA_IMG, args.LAMBDA_EDGE, d_f, d_v, d_g, args.NX, args.NY, args.NZ, args.NPROJ, args.NDX, args.NDY, args.HALFDET, args.PIXELSIZE, args.SOD, args.SDD, args.MAX_RAYLEN, args.SAMPLESIZE, args.HALFSIZE, args.ALPHA, args.BETA, args.EPSILON, debug.d_length);

		err = hipGetLastError();

		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to launch MumfordShah kernel (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
    }

	err=hipMemcpy(out.img, d_f, sizeof(float)*args.NX*args.NY*args.NZ, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector f from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err=hipMemcpy(out.edge, d_v, sizeof(float)*args.NX*args.NY*args.NZ, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector v from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	hipMemcpyFromSymbol(&max_numb, HIP_SYMBOL(d_max_numb), sizeof(int));
	hipMemcpyFromSymbol(&ave_numb, HIP_SYMBOL(d_ave_numb), sizeof(float));
	
    int64_t global_end = timer_s();
	ave_numb /= args.NPROJ*args.NDX*args.NDY;
    cout << "===========================================" <<endl;
    cout << boost::format("TOTAL time used = %1% seconds") % (global_end-global_start) <<endl;
    cout<< boost::format("actual max raylen = %1%, average raylen = %2%") % max_numb % ave_numb << endl;

	debug.print();
	err=hipFree(d_f);
    if (err != hipSuccess){
       	fprintf(stderr, "Failed to free vector d_f (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}

	err=hipFree(d_v);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free vector d_v (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}

	err=hipFree(d_g);
    if (err != hipSuccess){
       	fprintf(stderr, "Failed to free vector d_f (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}
    

}

int main(int argc, char** argv) {
    Parameter args;

    args.parse_config(argc, argv);
    args.print_options();

    CTInput in = CTInput(args);
    CTOutput out = CTOutput(args);

    in.read_sino(args.RAW_DATA_FILE);
    ct3d(args,in,out);
    out.write_img(args.OUTPUT_DIR);
    out.write_edge(args.OUTPUT_DIR);

    return 0;
}
